#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "ibgda_device.cuh"

namespace deep_ep {

// Forward declaration of ExpertSyncInfo from config.hpp
struct ExpertSyncInfo {
    int expected_tokens_per_rank[8];
    int received_tokens_per_rank[8];
    int total_expected_tokens;
    int total_received_tokens;
    int completed_ranks;
    int expert_processing_complete;
    void* combined_x_ptr;  // Pointer to combined_x buffer in NVSHMEM symmetric heap
    int padding[1];
};

namespace internode_ll {

// Diagnostic arrays for tracking sync points and block execution
__device__ int g_sync_counter[1024];      // Track progress state for each block
__device__ int g_sync_reached[1024];      // Track which blocks reached sync point
__device__ int g_active_blocks[1024];     // Track which blocks are actually processing
__device__ int g_block_expert_idx[1024];  // Track which expert each block is processing

template <int kNumThreads> __launch_bounds__(kNumThreads, 1)
__global__ void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                                         int* clean_1, int num_clean_int_1) {
    // Barrier before cleaning (in case of unfinished chunked EP)
    nvshmemx_barrier_all_block();

    // Clean - handle null pointers gracefully
    auto thread_id = static_cast<int>(threadIdx.x);
    if (clean_0 != nullptr) {
        #pragma unroll
        for (int i = thread_id; i < num_clean_int_0; i += kNumThreads)
            clean_0[i] = 0;
    }
    if (clean_1 != nullptr) {
        #pragma unroll
        for (int i = thread_id; i < num_clean_int_1; i += kNumThreads)
            clean_1[i] = 0;
    }

    // Barrier after cleaning (make sure the low-latency mode works fine)
    nvshmemx_barrier_all_block();
}

// Extended version that also cleans ExpertSyncInfo
template <int kNumThreads> __launch_bounds__(kNumThreads, 1)
__global__ void clean_low_latency_buffer_with_sync(int* clean_0, int num_clean_int_0,
                                                   int* clean_1, int num_clean_int_1,
                                                   ExpertSyncInfo* expert_sync_info,
                                                   int num_experts) {
    // Barrier before cleaning (in case of unfinished chunked EP)
    nvshmemx_barrier_all_block();

    // Clean - handle null pointers gracefully
    auto thread_id = static_cast<int>(threadIdx.x);
    if (clean_0 != nullptr) {
        #pragma unroll
        for (int i = thread_id; i < num_clean_int_0; i += kNumThreads)
            clean_0[i] = 0;
    }
    if (clean_1 != nullptr) {
        #pragma unroll
        for (int i = thread_id; i < num_clean_int_1; i += kNumThreads)
            clean_1[i] = 0;
    }

    // Clean ExpertSyncInfo
    if (expert_sync_info != nullptr) {
        for (int expert_idx = thread_id; expert_idx < num_experts; expert_idx += kNumThreads) {
            // Initialize all fields to zero
            for (int rank = 0; rank < 8; ++rank) {
                expert_sync_info[expert_idx].expected_tokens_per_rank[rank] = 0;
                expert_sync_info[expert_idx].received_tokens_per_rank[rank] = 0;
            }
            expert_sync_info[expert_idx].total_expected_tokens = 0;
            expert_sync_info[expert_idx].total_received_tokens = 0;
            expert_sync_info[expert_idx].completed_ranks = 0;
            expert_sync_info[expert_idx].expert_processing_complete = 0;
            for (int i = 0; i < 1; ++i) {
                expert_sync_info[expert_idx].padding[i] = 0;
            }
        }
    }

    // Barrier after cleaning (make sure the low-latency mode works fine)
    nvshmemx_barrier_all_block();
}

void clean_low_latency_buffer(int* clean_0, int num_clean_int_0,
                              int* clean_1, int num_clean_int_1,
                              hipStream_t stream) {
    constexpr int kNumThreads = 256;

    // Skip if nothing to clean
    if ((clean_0 == nullptr || num_clean_int_0 == 0) &&
        (clean_1 == nullptr || num_clean_int_1 == 0)) {
        return;
    }

    SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, clean_low_latency_buffer<kNumThreads>,
                  clean_0, num_clean_int_0, clean_1, num_clean_int_1);
}

void clean_low_latency_buffer_with_sync(int* clean_0, int num_clean_int_0,
                                       int* clean_1, int num_clean_int_1,
                                       ExpertSyncInfo* expert_sync_info,
                                       int num_experts,
                                       hipStream_t stream) {
    constexpr int kNumThreads = 256;

    // Skip if nothing to clean (but still need to clean sync info)
    if ((clean_0 == nullptr || num_clean_int_0 == 0) &&
        (clean_1 == nullptr || num_clean_int_1 == 0) &&
        expert_sync_info == nullptr) {
        return;
    }

    SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, clean_low_latency_buffer_with_sync<kNumThreads>,
                  clean_0, num_clean_int_0, clean_1, num_clean_int_1,
                  expert_sync_info, num_experts);
}

template <bool kUseFP8, bool kUseUE8M0, int kHidden>
__global__ __launch_bounds__(1024, 1) void
dispatch(void* packed_recv_x, void* packed_recv_x_scales,
         int* packed_recv_src_info, int64_t* packed_recv_layout_range,
         int* packed_recv_count,
         int* cumulative_local_expert_recv_stats,
         int64_t* dispatch_wait_recv_cost_stats,
         void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
         ExpertSyncInfo* expert_sync_info_buffer,
         void* combined_x,  // combined_x buffer for NVSHMEM GET
         const void* x, const int64_t* topk_idx,
         int* atomic_counter_per_expert, int* atomic_finish_counter_per_expert_unused,
         int* next_clean, int num_next_clean_int,
         const int* num_recv_tokens_per_rank,
         int num_tokens, int num_max_dispatch_tokens_per_rank,
         int num_topk, int num_experts, int rank, int num_ranks,
         int num_warp_groups, int num_warps_per_group,
         bool round_scale, int phases) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_local_experts = num_experts / num_ranks;

    // In Pure EP mode, num_ranks equals expert_parallel_size and there's no data parallelism
    // This means each rank processes ALL tokens but only for its local experts
    const bool is_pure_ep_mode = (num_ranks == num_experts / num_local_experts);

    // Initialize ExpertSyncInfo at the beginning of dispatch
    if (thread_id == 0 && sm_id == 0 && expert_sync_info_buffer != nullptr) {
        for (int i = 0; i < num_experts; i++) {
            // Initialize all fields to 0
            memset(&expert_sync_info_buffer[i], 0, sizeof(ExpertSyncInfo));
        }
        __threadfence_system();
    }
    __syncthreads();

    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = blockIdx.x;

    // Initialize diagnostic arrays at kernel start
    if (thread_id == 0) {
        g_sync_counter[sm_id] = 1;  // Block started
        g_block_expert_idx[sm_id] = responsible_expert_idx;
        g_sync_reached[sm_id] = 0;
        g_active_blocks[sm_id] = (responsible_expert_idx < num_experts) ? 1 : 0;
    }

    // May extract UE8M0 from the scales
    using scale_t = std::conditional_t<kUseUE8M0, uint8_t, float>;
    using packed_t = std::conditional_t<kUseUE8M0, uint32_t, float>;
    EP_STATIC_ASSERT(sizeof(packed_t) % sizeof(scale_t) == 0, "Invalid vector length");

    // FP8 staffs
    constexpr int kNumPerChannels = 128;
    const int num_scales = kHidden / kNumPerChannels;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));

    // Message package: hidden data, FP8 scales, index at source
    // NOTES: currently we have 3 reserved int fields for future use
    using vec_t = typename std::conditional<kUseFP8, int2, int4>::type;
    // Separate metadata size from data size for consistency
    const size_t num_bytes_per_data = kUseFP8 ? (kHidden + num_scales * sizeof(float)) : (kHidden * sizeof(hip_bfloat16));
    const size_t num_bytes_per_msg = sizeof(int4) + num_bytes_per_data;  // Metadata + data for RDMA
    const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
    const size_t num_int4_per_data = num_bytes_per_data / sizeof(int4);  // Data only size
    EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);
    EP_DEVICE_ASSERT(num_bytes_per_data % sizeof(int4) == 0);

    // Expert counts
    constexpr int kNumMaxWarpGroups = 32;
    __shared__ int shared_num_tokens_sent_per_expert[kNumMaxWarpGroups];

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0) {
        goto LOW_LATENCY_DISPATCH_RECV;
    }


    // NOTE: Finish counter initialization removed - not needed with simplified approach

    // There are 2 kinds of warps in this part:
    // 1. The first-kind warps for FP8 cast and sending top-k tokens
    // 2. The last warp for reading `topk_idx` and count for per-expert information
    if (warp_id < num_warps - 1) {
        constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(hip_bfloat16);
        EP_DEVICE_ASSERT(kHidden % kNumElemsPerRead == 0);
        EP_STATIC_ASSERT(kNumElemsPerRead * 32 % kNumPerChannels == 0, "Invalid vectorization");
        const size_t hidden_bf16_int4 = kHidden / kNumElemsPerRead;

        // Token processing: Each block processes ALL tokens but only sends to its responsible expert
        // This ensures every expert receives its assigned tokens regardless of block assignment


        // All warps except the last one participate in token processing
        // Distribute tokens across warps to prevent duplicate processing
        const int num_processing_warps = num_warps - 1;
        for (int token_idx = warp_id; token_idx < num_tokens; token_idx += num_processing_warps) {
            const auto x_int4 = static_cast<const int4*>(x) + token_idx * num_int4_per_data;
            const auto rdma_x_src_idx = reinterpret_cast<int*>(static_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
            const auto rdma_x_vec = reinterpret_cast<vec_t*>(reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
            const auto rdma_x_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

            // In Pure EP mode, we need to make this token available to ALL ranks
            // Each rank will process it with their local experts
            if (is_pure_ep_mode) {
                // Store token id header for this token (per-warp leader)
                if (lane_id == 0) {
                    *rdma_x_src_idx = token_idx;
                }
                // __syncwarp();
            } else {
                // Mixed mode: normal processing
                if (lane_id == 0) {
                    *rdma_x_src_idx = token_idx;
                }
                // __syncwarp();
            }

            // FP8 cast (do this once per token, not per expert)
            // All threads in the processing warp participate in FP8 cast
            #pragma unroll
            for (int i = lane_id; i < hidden_bf16_int4; i += 32) {
                // Read
                auto int4_value = __ldg(x_int4 + i);

                if constexpr (kUseFP8) {
                    // Calculate local amax
                    auto bf16_values = reinterpret_cast<hip_bfloat16*>(&int4_value);
                    float fp32_values[kNumElemsPerRead];
                    float amax = kFP8Margin, scale, scale_inv;
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; ++ j) {
                        fp32_values[j] = static_cast<float>(bf16_values[j]);
                        amax = fmaxf(amax, fabsf(fp32_values[j]));
                    }

                    // Reduce amax and scale
                    EP_STATIC_ASSERT(kNumElemsPerRead * 32 / kNumPerChannels == 2, "Invalid vectorization");
                    amax = half_warp_reduce_max(amax);
                    calculate_fp8_scales(amax, scale, scale_inv, round_scale);
                    if (lane_id == 0 or lane_id == 16)
                        rdma_x_scales[i * kNumElemsPerRead / 128] = scale_inv;

                    // Cast into send buffer
                    vec_t int2_value;
                    auto fp8x2_values = reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; j += 2) {
                        float2 fp32x2 = {fp32_values[j] * scale, fp32_values[j + 1] * scale};
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
                    }
                    rdma_x_vec[i] = int2_value;
                } else {
                    // Reinterpret-cast is for C++14 compatibility
                    rdma_x_vec[i] = *reinterpret_cast<vec_t*>(&int4_value);
                }
            }
            // Sync within the processing warp
            // __syncwarp();

            // Each block only processes tokens destined for its responsible expert
            // This prevents multiple blocks from incrementing the same expert's counter
            // Token ownership masking: In Pure EP, only the owner rank dispatches this token
            int owner_rank_for_token = is_pure_ep_mode ? (token_idx % num_ranks) : -1;

            for (int topk_offset = 0; topk_offset < num_topk; topk_offset++) {
                auto dst_expert_idx = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + topk_offset));

                // Skip invalid experts (e.g., padding tokens with -1)
                if (dst_expert_idx < 0) {
                    continue;
                }

                // Skip if this token is not for our responsible expert
                if (dst_expert_idx != responsible_expert_idx) {
                    continue;
                }

                // Ownership mask: only the owner rank dispatches in Pure EP mode
                if (is_pure_ep_mode && rank != owner_rank_for_token) {
                    continue;
                }

                // Issue IBGDA sends
                if (dst_expert_idx >= 0) {
                    const auto dst_rank = dst_expert_idx / num_local_experts;
                    const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;


                    // Use per-rank counter to match buffer layout
                    // Counter index = dst_expert_idx * num_ranks + source_rank
                    const int counter_idx = dst_expert_idx * num_ranks + rank;
                    int slot_idx = lane_id == 0 ? atomicAdd(atomic_counter_per_expert + counter_idx, 1) : 0;
                    slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);

                    // Add bounds check to prevent buffer overflow
                    if (slot_idx >= num_max_dispatch_tokens_per_rank) {
                        // All threads must hit the assert to ensure proper termination
                        // __syncwarp();
                        assert(false && "Buffer overflow: Token dispatch buffer is full!");
                    }

                    // Process valid slots only

                    // NEW: Update expected token count in ExpertSyncInfo
                    if (lane_id == 0 && expert_sync_info_buffer != nullptr) {
                        atomicAdd(&expert_sync_info_buffer[dst_expert_idx].expected_tokens_per_rank[rank], 1);
                        atomicAdd(&expert_sync_info_buffer[dst_expert_idx].total_expected_tokens, 1);
                    }

                    const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
                    const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                                            dst_expert_local_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                            rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                                            slot_idx * num_bytes_per_msg;
                    const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                    if (dst_p2p_ptr == 0) {
                        nvshmemi_ibgda_put_nbi_warp(dst_ptr, src_ptr, num_bytes_per_msg, dst_rank, dst_expert_local_idx, lane_id, slot_idx);
                    } else {
                        // NOTES: only 2 load iterations for 7K hidden with 8 unrolls
                        const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
                        const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_p2p_ptr);
                        UNROLLED_WARP_COPY(8, lane_id, num_int4_per_msg, dst_int4_ptr, src_int4_ptr, ld_nc_global, st_na_global);
                        // Enhanced synchronization for P2P write
                        // __syncwarp();
                        // __threadfence_system();
                        asm volatile("membar.sys;");  // Additional barrier for P2P
                    }

                    // Increase counter after finishing
                    // if (dst_p2p_ptr == 0) {
                    //     // For RDMA path, sync after the operation
                    //     __syncwarp();  // First sync warp
                    //     __threadfence_system();  // Then memory fence to ensure RDMA writes are visible
                    // }
                    // NOTE: Per-token finish counter update removed - will update once after all tokens sent
                }  // End of if (dst_expert_idx >= 0)
            }  // End of expert processing loop
        }  // End of token processing loop
    }

    if (warp_id == num_warps - 1) {

        if (sm_id == 0) {

            // The first SM is also responsible for cleaning the next buffer
            #pragma unroll
            for (int i = lane_id; i < num_next_clean_int; i += 32)
                next_clean[i] = 0;
        }

        // This SM should be responsible for one destination expert
        const auto expert_begin_idx = blockIdx.x;
        const auto expert_end_idx = min(expert_begin_idx + 1, num_experts);

        // Store the actual sent count for later use
        if (expert_begin_idx < num_experts && thread_id == 0) {
            const int my_counter_idx = expert_begin_idx * num_ranks + rank;
            int actual_sent_count = atomic_counter_per_expert[my_counter_idx];

            // Store the actual sent count for count send phase
            shared_num_tokens_sent_per_expert[0] = actual_sent_count;
        }
    }
    __syncthreads();

    // Issue count sends - each block handles its own expert
    // IMPORTANT: This section must complete quickly to avoid grid sync deadlock

    if (responsible_expert_idx < num_experts) {
        // Only one thread per block performs the count send
        if (sub_warp_id == 0 and lane_id == 0) {
        // The actual count is in atomic_counter_per_expert after all sends
        // This reflects tokens actually sent (excluding dropped ones)
        // Since we now use per-rank counters, need to sum all ranks
        // This SM processed tokens from current rank only
        const int my_counter_idx = responsible_expert_idx * num_ranks + rank;
        const auto num_tokens_sent = atomic_counter_per_expert[my_counter_idx];

        // Calculate expected count from ALL tokens
        // Each warp processes only its assigned tokens
        int expected_count = 0;

        // Check all tokens for this expert, but only count tokens processed by all warps
        for (int token_idx = 0; token_idx < num_tokens; token_idx++) {
            // Check top-k experts for this token
            for (int k = 0; k < num_topk; k++) {
                auto idx = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + k));
                // Skip invalid experts (padding tokens)
                if (idx < 0) {
                    continue;
                }
                if (idx == responsible_expert_idx) {
                    // Ownership masking: In Pure EP, only the owner rank sends this token
                    if (!is_pure_ep_mode) {
                        expected_count++;
                    } else {
                        const int owner_rank = token_idx % num_ranks;
                        if (owner_rank == rank) expected_count++;
                    }
                }
            }
        }

        // Check for token drops - MUST NEVER HAPPEN
        if (num_tokens_sent < expected_count) {
            printf("[FATAL ERROR] Token drop detected!\n");
            printf("  Expert %d: Sent only %d tokens (expected %d, dropped %d)\n",
                   responsible_expert_idx, num_tokens_sent, expected_count, expected_count - num_tokens_sent);
            printf("  This is a critical error that must be fixed\n");
            printf("  Block: %d, Rank: %d, num_warps: %d, num_topk: %d\n", sm_id, rank, num_warps, num_topk);
            assert(false && "Token drop detected! This must never happen.");
        }

        // Count send moved to after grid sync to prevent deadlock

        // Note: packed_recv_count is already initialized to zero in deep_ep.cpp (line 1123)
        // No need to initialize it here in the kernel
        }  // End of if (sub_warp_id == 0 and lane_id == 0)
    }  // End of if (responsible_expert_idx < num_experts) - moved brace here

    // Ensure all threads in the block complete before moving on
    // This must happen OUTSIDE the if statement to ensure ALL blocks sync
    __syncthreads();

    if (responsible_expert_idx >= num_experts) {
        if (thread_id == 0) {
            g_active_blocks[sm_id] = 0;  // Mark as inactive
        }
    }

    // Ensure ALL blocks reach this point before grid sync
    // This barrier ensures that both active and inactive blocks are at the same point
    __syncthreads();



    // Track if this block actually performed any NVSHMEM puts
    __shared__ int block_put_count;
    if (thread_id == 0) {
        if (responsible_expert_idx < num_experts) {
            const int my_counter_idx = responsible_expert_idx * num_ranks + rank;
            block_put_count = atomic_counter_per_expert[my_counter_idx];
        } else {
            block_put_count = 0;
        }
    }
    __syncthreads();

    // Don't use nvshmem_quiet() here - it causes deadlock
    // Grid sync is sufficient to ensure all blocks complete
    __syncthreads();  // Ensure all threads wait

    // Grid sync after send phase to ensure all ranks complete sending
    // This must happen before any rank checks if it should skip receive phase
    if (phases & LOW_LATENCY_SEND_PHASE) {
        // Update sync counter before sync
        if (thread_id == 0) {
            g_sync_counter[sm_id] = 2;  // Approaching sync
            g_sync_reached[sm_id] = 1;
        }

        __syncthreads();  // Block-wide sync first

        // Check for conditional sync skip
        #ifdef DEEPEP_SKIP_GRID_SYNC
        #else
            cg::this_grid().sync();  // Then grid-wide sync

            if (thread_id == 0) {
                g_sync_counter[sm_id] = 3;  // Passed sync
            }
        #endif

        __threadfence_system();  // Ensure all writes are visible
    }

    // Send counts after dispatch send phase completes
    // This MUST be outside the send phase block to handle return_recv_hook case
    // When return_recv_hook=true, dispatch only runs send phase and exits
    // Count must still be sent for later combine kernel to work correctly
    //
    // The rdma_recv_count buffer MUST NOT be cleared between dispatch
    // send and receive phases! The clean_low_latency_buffer function now excludes
    // the count buffer to preserve these values.
    //
        // Always send counts regardless of phase to prevent deadlock
        // All ranks must send counts, even if they only run RECV_PHASE
        // EACH BLOCK sends count for its responsible expert only
    if (responsible_expert_idx < num_experts && thread_id == 0 && atomic_counter_per_expert != nullptr) {
        const int dst_rank = responsible_expert_idx / num_local_experts;
        const int dst_expert_local_idx = responsible_expert_idx % num_local_experts;

        // This is the current rank's count for this expert
        const int counter_idx = responsible_expert_idx * num_ranks + rank;
        int actual_sent_count = atomic_counter_per_expert[counter_idx];

        // Ownership masking: In Pure EP, ensure non-owner ranks report zero for non-owned tokens
        if (is_pure_ep_mode) {
            // NOTE: atomic_counter_per_expert는 누적 카운터이므로, 여기서는 음수 카운트 프로토콜(-n-1)을 유지하면서
            //       비-오너 랭크가 보낸 토큰이 없도록 위에서 디스패치가 이미 차단되어야 합니다.
            //       혹시라도 누수가 있으면 카운트를 0으로 강제하여 리시브 측 프로토콜 일관성을 보장합니다.
            if (actual_sent_count < 0) actual_sent_count = 0;
        }

        // We need to write to the destination rank's buffer
        // The count buffer is arranged as [num_local_experts][num_ranks]
        // We want to write to dst_rank's buffer at position [dst_expert_local_idx][src_rank]
        // where src_rank is us (rank)

        // Calculate the offset in the destination rank's count buffer
        // dst_expert_local_idx: which local expert on the destination rank
        // rank: our rank (the source rank from dst's perspective)
        int dst_offset = dst_expert_local_idx * num_ranks + rank;

        // Get the NVSHMEM pointer for the destination rank's count buffer
        // rdma_recv_count is in the NVSHMEM symmetric heap
        auto dst_addr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_offset);
        auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_addr, rank, dst_rank);

        if (dst_p2p_ptr == 0) {
            // Use IBGDA for remote access
            #ifdef DEEPEP_VERBOSE_DEBUG
            printf("[COUNT_SEND_IBGDA] rank=%d block=%d -> rank=%d: sending count=%d to expert=%d (offset=%d)",
                   rank, sm_id, dst_rank, -actual_sent_count - 1, dst_expert_local_idx, dst_offset);
            #endif
            nvshmemi_ibgda_amo_nonfetch_add(rdma_recv_count + dst_offset,
                -actual_sent_count - 1, dst_rank, dst_expert_local_idx);
        } else {
            // Use atomic for P2P path
            #ifdef DEEPEP_VERBOSE_DEBUG
            printf("[COUNT_SEND_P2P] rank=%d block=%d -> rank=%d: sending count=%d to expert=%d (offset=%d)",
                   rank, sm_id, dst_rank, -actual_sent_count - 1, dst_expert_local_idx, dst_offset);
            #endif
            atomicAdd_system(reinterpret_cast<int*>(dst_p2p_ptr),
                -actual_sent_count - 1);
        }

        // Ensure count send is visible before proceeding
        __threadfence_system();
    }

    // Ensure all blocks have sent their counts before any block starts receiving
    // This prevents the timeout issue where blocks wait for counts that haven't been sent yet
    __syncthreads();  // Block-wide sync first
    cg::this_grid().sync();  // Grid-wide sync to ensure all counts are sent
    __threadfence_system();  // Ensure all count writes are globally visible


    // Receiving phase
    LOW_LATENCY_DISPATCH_RECV:

    if ((phases & LOW_LATENCY_RECV_PHASE) == 0) {
        return;
    }


    // Determine if this rank should process tokens
    bool should_process = (num_recv_tokens_per_rank[rank] > 0);

    // Don't return early! Even if this rank doesn't receive tokens,
    // blocks still need to participate in the receive protocol
    // The early return was causing some blocks to never execute their receive logic
    // which could cause hangs in other ranks waiting for responses

    // Receiving and packing
    // In dispatch receive, we need to receive data sent TO our local experts FROM other ranks
    // Each block processes one (local_expert, src_rank) pair
    if (should_process && responsible_expert_idx < num_local_experts * num_ranks) {
        const int pair_idx = responsible_expert_idx;
        const auto local_expert_idx = pair_idx / num_ranks;  // Which of OUR local experts
        const auto src_rank = pair_idx % num_ranks;  // From which rank

        // Multi-node validation: ensure expert indices are within valid range
        EP_DEVICE_ASSERT(local_expert_idx < num_local_experts);
        EP_DEVICE_ASSERT(src_rank < num_ranks);

        const auto rdma_recv_x_uint8 = static_cast<uint8_t*>(rdma_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                src_rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg;
        // Use data-only size for consistency with buffer layout
        const auto recv_src_info = packed_recv_src_info +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank +
                src_rank * num_max_dispatch_tokens_per_rank;
        const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;
        const auto num_aligned_scales = align<int>(num_scales, sizeof(float) / sizeof(scale_t));
        const auto recv_x_scales = static_cast<scale_t*>(packed_recv_x_scales) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_aligned_scales;

        // Shared between sub-warps in warp groups
        __shared__ int shared_num_recv_tokens[kNumMaxWarpGroups], shared_recv_token_begin_idx[kNumMaxWarpGroups];

        // Wait tokens to arrive
        // using sub-warp 1 to overlap with sub-warp 0
        int num_recv_tokens, recv_token_begin_idx;
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 15);
        if (sub_warp_id == 1 and lane_id == 0) {
            // Wait for count to arrive
            // Count protocol: -actual_count - 1 (so -1 means 0 tokens)
            num_recv_tokens = 0;
            int wait_iterations = 0;
            const int MAX_WAIT = 100000000;  // 100M iterations

            // In Pure EP mode, we must wait for counts from ALL ranks
            // because padding tokens might cause some experts to receive 0 tokens
            // but the count (-1) must still be sent and received
            while (true) {
                num_recv_tokens = ld_acquire_sys_global(rdma_recv_count + local_expert_idx * num_ranks + src_rank);

                // Check if count has arrived (any non-zero value)
                if (num_recv_tokens != 0) {
                    // Count received! Break out of wait loop
                    break;
                }

                // Still waiting...
                wait_iterations++;
                // Timeout check - but this should not happen in correct operation
                if (wait_iterations > MAX_WAIT) {
                    printf("[FATAL] rank=%d block=%d timeout waiting for count! local_expert=%d, src_rank=%d\n",
                           rank, blockIdx.x, local_expert_idx, src_rank);
                    printf("[FATAL] Expected count at rdma_recv_count[%d], but value is still %d\n",
                           local_expert_idx * num_ranks + src_rank, num_recv_tokens);
                    printf("[FATAL] This indicates src_rank=%d never sent count to dst_rank=%d for expert=%d\n",
                           src_rank, rank, local_expert_idx + rank * num_local_experts);
                    assert(false && "Timeout waiting for count - synchronization failure");
                }
            }

            // Process only if we have tokens
            if (num_recv_tokens != 0) {
                num_recv_tokens = -num_recv_tokens - 1;

            // Buffer layout is [local_expert][from_rank][token]
            // Each source rank has its own dedicated range in the buffer
            // We need a per-rank counter, not a global counter per expert
            // Use a temporary atomic counter in packed_recv_count for each (expert, rank) pair
            const int counter_idx = local_expert_idx * num_ranks + src_rank;
            recv_token_begin_idx = atomicAdd(packed_recv_count + counter_idx, num_recv_tokens);

            // Ensure we don't overflow the per-rank section of the buffer
            if (recv_token_begin_idx + num_recv_tokens > num_max_dispatch_tokens_per_rank) {
                // Assert instead of clamping to prevent silent data loss
                printf("[FATAL ERROR] Receive buffer overflow detected!\n");
                printf("  Rank: %d\n", rank);
                printf("  Local expert: %d (global expert %d)\n", local_expert_idx, rank * num_local_experts + local_expert_idx);
                printf("  Source rank: %d\n", src_rank);
                printf("  Receive begin index: %d\n", recv_token_begin_idx);
                printf("  Number of tokens to receive: %d\n", num_recv_tokens);
                printf("  Max tokens per rank: %d\n", num_max_dispatch_tokens_per_rank);
                printf("  Total would be: %d (exceeds buffer capacity)\n", recv_token_begin_idx + num_recv_tokens);
                printf("  packed_recv_count[%d] was: %d before this addition\n",
                       local_expert_idx, recv_token_begin_idx);

                // Force an assertion failure
                assert(false && "Receive buffer overflow: Too many tokens for expert! Increase num_max_dispatch_tokens_per_rank.");
            }

            shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
            shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
            recv_range[src_rank] = pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);
            if (cumulative_local_expert_recv_stats != nullptr)
                atomicAdd(cumulative_local_expert_recv_stats + local_expert_idx, num_recv_tokens);
            } else {
                // No tokens for this (expert, rank) pair
                shared_num_recv_tokens[warp_group_id] = 0;
                shared_recv_token_begin_idx[warp_group_id] = 0;
                if (lane_id == 0) {
                    printf("[RECV SKIP] Block %d: No tokens for local_expert=%d from src_rank=%d\n",
                           blockIdx.x, local_expert_idx, src_rank);
                }
            }
        }
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 2), "r"(num_warps_per_group * 32));
        num_recv_tokens = shared_num_recv_tokens[warp_group_id];
        recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];

        // Skip token copying if no tokens
        if (num_recv_tokens == 0) {
            return;
        }

        // Copy tokens
        EP_DEVICE_ASSERT(num_scales <= 64);
        for (int i = sub_warp_id; i < num_recv_tokens; i += num_warps_per_group) {
            // Copy source info
            const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * num_bytes_per_msg);
            if (lane_id == 0)
                // Now recv_src_info already points to the correct rank's section
                recv_src_info[i] = ld_nc_global(src_src_idx);
            // __syncwarp();

            // Copy data
            // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
            const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));

            // Calculate destination address - packed_recv_x is just data, no metadata
            // Buffer layout: [local_expert][from_rank][token][hidden]
            // Use consistent data-only size (num_int4_per_data)

            // Verify offset calculation doesn't exceed buffer bounds
            const int token_slot_idx = recv_token_begin_idx + i;
            if (token_slot_idx >= num_max_dispatch_tokens_per_rank) {
                if (lane_id == 0) {
                    printf("[FATAL] Token slot index %d exceeds buffer capacity %d!\n",
                           token_slot_idx, num_max_dispatch_tokens_per_rank);
                    printf("  recv_token_begin_idx=%d, i=%d, local_expert=%d, src_rank=%d\n",
                           recv_token_begin_idx, i, local_expert_idx, src_rank);
                    assert(false && "Token slot index exceeds buffer bounds!");
                }
            }

            const auto dst_data = static_cast<int4*>(packed_recv_x) +
                local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_int4_per_data +
                src_rank * num_max_dispatch_tokens_per_rank * num_int4_per_data +
                token_slot_idx * num_int4_per_data;

            // Use data-only size for copy
            UNROLLED_WARP_COPY(7, lane_id, num_int4_per_data, dst_data, src_data, ld_nc_global, st_na_global);
            // Copy scales
            if constexpr (kUseFP8) {
                // Equivalent CuTe layout:
                //   (num_tokens, (num_packed, num_elems_per_pack)):(num_elems_per_pack, (num_tokens * num_elems_per_pack, 1))
                const auto src_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
                const auto num_elems_per_pack = static_cast<int>(sizeof(packed_t) / sizeof(scale_t));
                const auto token_idx = recv_token_begin_idx + i;
                const auto token_stride = num_elems_per_pack;
                const auto pack_stride = num_ranks * num_max_dispatch_tokens_per_rank * num_elems_per_pack;
                if (lane_id < num_scales) {
                    const auto pack_idx = lane_id / num_elems_per_pack;
                    const auto elem_idx = lane_id % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(ld_nc_global(src_scales + lane_id));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
                if (lane_id + 32 < num_scales) {
                    const auto pack_idx = (lane_id + 32) / num_elems_per_pack;
                    const auto elem_idx = (lane_id + 32) % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(ld_nc_global(src_scales + lane_id + 32));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
            }
        }

        // Add system-wide memory fence to ensure all writes are visible
        // This is critical for cross-node RDMA operations
        // All threads must execute the fence for consistency
        __threadfence_system();
    }

    // Final diagnostic: kernel exit
    if (thread_id == 0) {
        g_sync_counter[sm_id] = 4;  // Kernel completed
    }
}

void dispatch(void* packed_recv_x, void* packed_recv_x_scales,
              int* packed_recv_src_info, int64_t* packed_recv_layout_range,
              int* packed_recv_count,
              int* cumulative_local_expert_recv_stats,
              int64_t* dispatch_wait_recv_cost_stats,
              void* rdma_recv_x, int* rdma_recv_count, void* rdma_x,
              ExpertSyncInfo* expert_sync_info_buffer,
              void* combined_x,  // NEW: combined_x buffer for NVSHMEM GET
              const void* x, const int64_t* topk_idx,
              int* next_clean, int num_next_clean_int,
              const int* num_recv_tokens_per_rank,
              int num_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
              int num_topk, int num_experts, int rank, int num_ranks,
              bool use_fp8, bool round_scale, bool use_ue8m0,
              void* workspace, int num_device_sms,
              hipStream_t stream, int phases) {
    constexpr int kNumMaxTopK = 9;

    // Ensure we have enough warps to handle all top-k experts
    // We need at least num_topk + 1 warps (num_topk for processing, 1 for counting)
    const int min_warps_needed = num_topk + 1;

    // Calculate num_warp_groups based on the constraint that we need enough warps
    int num_warp_groups = 1;
    int num_warps_per_group = 32;

    // Find the best configuration
    for (int wg = 1; wg <= 32; wg++) {
        int wpg = 32 / wg;
        if (wpg > 0 && wg * wpg >= min_warps_needed && wg <= ceil_div(num_experts, num_device_sms)) {
            num_warp_groups = wg;
            num_warps_per_group = wpg;
            break;
        }
    }

    // Verify we have enough warps
    const auto num_warps = num_warp_groups * num_warps_per_group;
    if (num_warps < min_warps_needed) {
        printf("FATAL: Cannot allocate enough warps! Need %d warps but only have %d\n",
               min_warps_needed, num_warps);
        printf("  num_topk=%d, num_experts=%d, num_device_sms=%d\n",
               num_topk, num_experts, num_device_sms);
        EP_HOST_ASSERT(false && "Insufficient warps for top-k processing");
    }

    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);
    EP_HOST_ASSERT(kNumMaxTopK + 1 <= num_warp_groups * num_warps_per_group);

    const auto num_sms = ceil_div(num_experts, num_warp_groups);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

    // Allocate workspace for per-rank counters
    // Layout: [num_experts][num_ranks] for both counter types
    const int counters_per_type = num_experts * num_ranks;
    auto atomic_counter_per_expert_rank = static_cast<int*>(workspace);
    auto atomic_finish_counter_per_expert_rank = atomic_counter_per_expert_rank + counters_per_type;
    // For backward compatibility, also create pointers to per-expert counters
    auto atomic_counter_per_expert = atomic_counter_per_expert_rank;  // Will be used with rank offset
    auto atomic_finish_counter_per_expert_unused = atomic_finish_counter_per_expert_rank;  // Not used anymore
    EP_HOST_ASSERT(counters_per_type * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);

    // FP8 checks
    if (use_ue8m0)
        EP_HOST_ASSERT(round_scale and "UE8M0 SF requires `round_scale=True`");

#define DISPATCH_LAUNCH_CASE(hidden) { \
auto dispatch_func = dispatch<false, false, hidden>; \
if (use_fp8 and not use_ue8m0) \
    dispatch_func = dispatch<true, false, hidden>; \
if (use_fp8 and use_ue8m0) \
    dispatch_func = dispatch<true, true, hidden>; \
LAUNCH_KERNEL(&cfg, dispatch_func, \
              packed_recv_x, packed_recv_x_scales, \
              packed_recv_src_info, packed_recv_layout_range, \
              packed_recv_count, \
              cumulative_local_expert_recv_stats, \
              dispatch_wait_recv_cost_stats, \
              rdma_recv_x, rdma_recv_count, rdma_x, \
              expert_sync_info_buffer, \
              combined_x, \
              x, topk_idx, \
              atomic_counter_per_expert, atomic_finish_counter_per_expert_unused, \
              next_clean, num_next_clean_int, \
              num_recv_tokens_per_rank, \
              num_tokens, num_max_dispatch_tokens_per_rank, \
              num_topk, num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              round_scale, phases); } break

    // Set grid size to num_experts to ensure all experts are processed
    // Each block will handle one expert
    const int dispatch_grid_size = num_experts;
    SETUP_LAUNCH_CONFIG(dispatch_grid_size, num_warps * 32, stream);
    SWITCH_HIDDEN(DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <int kHidden, int kNumMaxTopk>
__global__ __launch_bounds__(1024, 1) void
combine(void* combined_x,
        void* fp32_workspace,  // Separate FP32 workspace for NVSHMEM reduction
        void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
        const void* x, const int64_t* topk_idx, const float* topk_weights,
        const int* src_info, const int64_t* layout_range,
        int64_t* combine_wait_recv_cost_stats,
        int* next_clean, int num_next_clean_int,
        int* atomic_clean_flag,
        ExpertSyncInfo* expert_sync_info_buffer,
        int num_combined_tokens, int hidden, int num_topk,
        int num_max_dispatch_tokens_per_rank,
        int num_experts, int rank, int num_ranks,
        int num_warp_groups, int num_warps_per_group,
        int phases, bool zero_copy) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    // Use gridDim.x which equals num_experts from the launch config
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto num_threads = static_cast<int>(blockDim.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;

    // Store num_topk in shared memory immediately to prevent corruption
    __shared__ int shared_num_topk;
    __shared__ int shared_num_experts;
    __shared__ int shared_num_ranks;
    __shared__ int shared_num_combined_tokens;
    __shared__ int shared_hidden;
    __shared__ int shared_num_max_dispatch_tokens_per_rank;

    if (thread_id == 0) {
        shared_num_topk = num_topk;
        shared_num_experts = num_experts;
        shared_num_ranks = num_ranks;
        shared_num_combined_tokens = num_combined_tokens;
        shared_hidden = hidden;
        shared_num_max_dispatch_tokens_per_rank = num_max_dispatch_tokens_per_rank;
    }
    __syncthreads();

    // Use shared memory values from now on
    int safe_num_topk = shared_num_topk;
    int safe_num_experts = shared_num_experts;
    int safe_num_ranks = shared_num_ranks;
    int safe_num_combined_tokens = shared_num_combined_tokens;
    int safe_hidden = shared_hidden;
    int safe_num_max_dispatch_tokens_per_rank = shared_num_max_dispatch_tokens_per_rank;
    int safe_num_local_experts = safe_num_experts / safe_num_ranks;

    //  Detect Pure EP mode
    const bool is_pure_ep_mode = (safe_num_ranks == safe_num_experts / safe_num_local_experts);

    // blockIdx.x is now limited by grid_size (num_device_sms), not num_experts
    const auto responsible_expert_idx = blockIdx.x;

    // Data type staffs
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);

    // Use actual hidden size from safe parameter, not template parameter
    const size_t actual_num_bytes_per_data = safe_hidden * sizeof(hip_bfloat16);
    const size_t actual_num_int4_per_data = actual_num_bytes_per_data / sizeof(int4);

    // Verify alignment
    EP_DEVICE_ASSERT(safe_hidden % kNumElemsPerInt4 == 0);
    EP_DEVICE_ASSERT(actual_num_bytes_per_data % sizeof(int4) == 0);

    // For compile-time checks, keep template-based calculations
    constexpr size_t num_bytes_per_data = kHidden * sizeof(hip_bfloat16);
    constexpr size_t num_bytes_per_msg = sizeof(int4) + num_bytes_per_data;
    EP_STATIC_ASSERT(num_bytes_per_data % sizeof(int4) == 0, "Invalid vectorization");
    EP_STATIC_ASSERT(num_bytes_per_msg % sizeof(int4) == 0, "Invalid message size");

    // dispatch uses num_bytes_per_msg for RDMA transfers (includes metadata)
    // But packed_recv_x buffer uses data-only layout

    // Count sending must be done in dispatch kernel, not combine
    // The combine kernel doesn't have access to rdma_recv_count buffer
    // Count sending was moved to dispatch kernel to fix this issue

    // Declare expert_idx before goto to avoid initialization bypass
    int expert_idx;

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_COMBINE_RECV;

    // Clean up next buffer
    if (sm_id == 0 and warp_group_id == 0 and sub_warp_id == 0) {
        #pragma unroll
        for (int i = lane_id; i < num_next_clean_int; i += 32)
            next_clean[i] = 0;

        // Notify before executing `int_p`
        // __syncwarp();
        if (lane_id == 0)
            atomic_add_release_global(atomic_clean_flag, num_experts);
    }

    // Issue IBGDA sends
    // Each block processes its assigned expert
    expert_idx = responsible_expert_idx;
    if (expert_idx < num_experts) {
        const auto dst_rank = expert_idx / num_local_experts;
        const auto local_expert_idx = expert_idx % num_local_experts;
        const auto layout = __ldg(layout_range + local_expert_idx * num_ranks + dst_rank);
        const auto local_x = static_cast<const int4*>(x) +
                local_expert_idx * safe_num_ranks * safe_num_max_dispatch_tokens_per_rank * actual_num_int4_per_data;
        const auto local_src_info = src_info + local_expert_idx * safe_num_ranks * safe_num_max_dispatch_tokens_per_rank;
        const auto rdma_send_x_vec = static_cast<uint8_t*>(rdma_send_x) +
                local_expert_idx * safe_num_ranks * safe_num_max_dispatch_tokens_per_rank * num_bytes_per_data;

        // Unpack layout
        int offset = 0, num_tokens_to_send = 0;
        unpack2(layout, num_tokens_to_send, offset);

        // Issue IBGDA send
        for (int token_idx = offset + sub_warp_id; token_idx < offset + num_tokens_to_send; token_idx += num_warps_per_group) {
            // local_x already points to this expert's data, need to add rank offset
            const auto x_int4 = local_x + dst_rank * safe_num_max_dispatch_tokens_per_rank * actual_num_int4_per_data + token_idx * actual_num_int4_per_data;
            // Add rank offset for rdma_send_x_vec access
            const auto rdma_send_type_row = reinterpret_cast<int*>(rdma_send_x_vec + dst_rank * safe_num_max_dispatch_tokens_per_rank * num_bytes_per_data + token_idx * num_bytes_per_data);
            const auto rdma_send_x_vec_row = reinterpret_cast<uint8_t*>(rdma_send_type_row);

            // Copy directly to local rank, or copy to buffer and issue RDMA
            // Add rank offset for src_info access
            const auto src_idx = __ldg(local_src_info + dst_rank * safe_num_max_dispatch_tokens_per_rank + token_idx);
            const auto buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);

            // Calculate slot_idx from token_idx - offset
            // token_idx ranges from offset to offset+num_tokens_to_send-1
            // slot_idx should range from 0 to num_tokens_to_send-1
            const auto slot_idx = token_idx - offset;

            // Bounds check for slot_idx
            if (slot_idx < 0 || slot_idx >= safe_num_max_dispatch_tokens_per_rank) {
                if (lane_id == 0) {
                    printf("[ERROR] combine kernel: slot_idx out of bounds! token_idx=%d, offset=%d, slot_idx=%d, max=%d",
                           token_idx, offset, slot_idx, safe_num_max_dispatch_tokens_per_rank);
                }
                continue;
            }
            // Use local expert index and rank-based offset calculation to match dispatch buffer layout
            // Must use num_bytes_per_msg to match dispatch's buffer layout
            const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                                 (local_expert_idx * safe_num_ranks * safe_num_max_dispatch_tokens_per_rank +
                                  rank * safe_num_max_dispatch_tokens_per_rank +
                                  slot_idx) * num_bytes_per_msg;
            const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            if (dst_p2p_ptr == 0) {
                // IBGDA path: prepare a contiguous message with [header(int4) | data]
                // 1) Copy data right after header space in the local send buffer
                const auto buf_int4_ptr = reinterpret_cast<int4*>(buf_ptr);
                if (not zero_copy)
                    UNROLLED_WARP_COPY(7, lane_id, actual_num_int4_per_data, buf_int4_ptr + 1, x_int4, ld_nc_global, st_na_global);

                // 2) Write header (source token index) into the first int of the header int4
                if (lane_id == 0) {
                    reinterpret_cast<int*>(rdma_send_x_vec_row)[0] = src_idx;
                }
                // __syncwarp();

                // 3) Send header then payload
                const auto dst_expert_local_idx = expert_idx % num_local_experts;
                // send header (int4 = 16 bytes)
                nvshmemi_ibgda_put_nbi_warp(dst_ptr, buf_ptr, sizeof(int4), dst_rank, dst_expert_local_idx, lane_id, slot_idx);
                // send data (bf16 payload)
                nvshmemi_ibgda_put_nbi_warp(dst_ptr + sizeof(int4), buf_ptr + sizeof(int4), hidden * sizeof(hip_bfloat16), dst_rank, dst_expert_local_idx, lane_id, slot_idx);
            } else {
                // P2P path: write header then data directly to remote mapped memory
                if (lane_id == 0) {
                    st_na_global(reinterpret_cast<int*>(dst_p2p_ptr), src_idx);
                }
                // __syncwarp();
                const auto dst_int4_ptr = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(dst_p2p_ptr) + sizeof(int4));
                UNROLLED_WARP_COPY(7, lane_id, actual_num_int4_per_data, dst_int4_ptr, x_int4, ld_nc_global, st_na_global);
                // Ensure visibility before flag set
                // __syncwarp();
                // __threadfence_system();
                asm volatile("membar.sys;");
            }
        }

        // Put the finishing flag
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 16);
        asm volatile("bar.sync %0, %1;" :: "r"(warp_group_id + 1), "r"(num_warps_per_group * 32));

        // Ensure ALL P2P writes from ALL warps complete before ANY flag is set
        __syncthreads();  // Block-wide synchronization

        if (num_tokens_to_send > 0 && sub_warp_id == 1 && lane_id == 0) {
            // Wait for all local processing to complete
            while (ld_acquire_global(atomic_clean_flag) == 0);

            // Use more robust synchronization
            // First, ensure all data writes are globally visible
            __threadfence_system();

            // Memory barrier to guarantee ordering
            asm volatile("membar.sys;");

            // Now update the flag with proper synchronization
            // Use dst_expert_local_idx for the destination's flag buffer
            const auto dst_expert_local_idx = expert_idx % num_local_experts;
            auto flag_dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_flag + dst_expert_local_idx);
            auto flag_dst_p2p_ptr = nvshmemi_get_p2p_ptr(flag_dst_ptr, rank, dst_rank);

            // Use correct QP ID for remote operations
            if (flag_dst_p2p_ptr == 0) {
                // For remote updates, use NVSHMEM atomic with destination's local expert index as QP ID
                nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(flag_dst_ptr), 1, dst_rank, dst_expert_local_idx);
            } else {
                // For P2P, use atomic with memory ordering
                atomicAdd_system(reinterpret_cast<int*>(flag_dst_p2p_ptr), 1);
            }

            // Update received token count in ExpertSyncInfo
            if (expert_sync_info_buffer != nullptr) {
                // Update the destination expert's received count
                atomicAdd(&expert_sync_info_buffer[expert_idx].received_tokens_per_rank[rank], num_tokens_to_send);
                atomicAdd(&expert_sync_info_buffer[expert_idx].total_received_tokens, num_tokens_to_send);
            }

            // Final memory fence to ensure flag update is visible
            __threadfence_system();
            asm volatile("membar.sys;");

            #ifdef DEEPEP_VERBOSE_DEBUG
            printf("[FLAG SET] rank=%d -> rank=%d, expert=%d, local_expert_idx=%d, blockIdx.x=%d\n",
                   rank, dst_rank, expert_idx, local_expert_idx, blockIdx.x);
            #endif
        }
        // __syncwarp();

        // Mark expert processing as complete
        if (warp_group_id == 0 && sub_warp_id == 0 && lane_id == 0 && expert_sync_info_buffer != nullptr) {
            atomicAdd(&expert_sync_info_buffer[expert_idx].expert_processing_complete, 1);
            __threadfence_system();
        }
    } // End of if (expert_idx < num_experts)

    // Ensure all expert processing is complete before starting combine
    __syncthreads();

    // Receiving phase
    LOW_LATENCY_COMBINE_RECV:
    __syncthreads();

    // Set flag to skip processing but continue to grid sync
    bool skip_recv_phase = ((phases & LOW_LATENCY_RECV_PHASE) == 0);
    if (skip_recv_phase) {
        if (thread_id == 0 && sm_id == 0) {
            printf("[COMBINE] rank=%d will skip combine processing (no RECV_PHASE in phases=0x%x) but participate in sync\n", rank, phases);
        }
    }

    // Each block waits for its assigned expert
    expert_idx = responsible_expert_idx;
    if (expert_idx < num_experts) {
        const auto expert_rank = expert_idx / num_local_experts;

        // Only process experts that belong to this rank
        if (expert_rank == rank) {
            const auto local_expert_idx = expert_idx % num_local_experts;

            // Check if any rank will send data to this expert by checking layout_range
            bool will_receive_data = false;
            if (layout_range != nullptr) {
                for (int src_rank = 0; src_rank < num_ranks; src_rank++) {
                    auto layout = __ldg(layout_range + local_expert_idx * num_ranks + src_rank);
                    int num_tokens_from_rank, offset;
                    unpack2(layout, num_tokens_from_rank, offset);
                    if (num_tokens_from_rank > 0) {
                        will_receive_data = true;
                        break;
                    }
                }
            }

            EP_DEVICE_ASSERT(num_warps_per_group > 1);
            if (sub_warp_id == 0 && lane_id == 0) {
                if (!will_receive_data) {
                    // No data expected for this expert - skip waiting and mark as complete
                    #ifdef DEEPEP_VERBOSE_DEBUG
                    printf("[FLAG SKIP] rank=%d expert=%d has no incoming data, skipping wait\n", rank, expert_idx);
                    #endif
                    // DO NOT return here! Must participate in grid sync below
                } else {
                    // se same address calculation as flag setting
                    // Flag setting uses local_expert_idx, so flag waiting must also use local_expert_idx
                    volatile int* flag_ptr = rdma_recv_flag + local_expert_idx;

                    #ifdef DEEPEP_VERBOSE_DEBUG
                    printf("[FLAG WAIT] rank=%d waiting for expert=%d (local_idx=%d) flag (expecting data)\n",
                           rank, expert_idx, local_expert_idx);
                    #endif

                    // Use ExpertSyncInfo for more robust synchronization
                    if (expert_sync_info_buffer != nullptr) {
                        ExpertSyncInfo* sync_info = &expert_sync_info_buffer[expert_idx];

                        // Wait until all expected tokens are received
                        int timeout_counter = 0;
                        while (sync_info->total_received_tokens < sync_info->total_expected_tokens) {
                            __threadfence_system();
                            if (++timeout_counter > 100000000) {
                                printf("[GPU HANG DETECTED] Rank %d waiting for expert %d (local_idx=%d)\n",
                                       rank, expert_idx, local_expert_idx);
                                printf("  Expected: %d tokens, Received: %d tokens\n",
                                       sync_info->total_expected_tokens, sync_info->total_received_tokens);
                                printf("  Per-rank breakdown:\n");
                                for (int r = 0; r < num_ranks && r < 8; ++r) {
                                    printf("    Rank %d: Expected=%d, Received=%d\n", r,
                                           sync_info->expected_tokens_per_rank[r],
                                           sync_info->received_tokens_per_rank[r]);
                                }
                                timeout_counter = 0;
                            }
                        }
                    } else {
                        // Fallback to old flag-based synchronization
                        int timeout_counter = 0;
                        while (ld_acquire_sys_global((const int*)flag_ptr) == 0) {
                            if (++timeout_counter > 100000000) {
                                printf("[GPU HANG DETECTED] Rank %d waiting for expert %d (local_idx=%d) flag indefinitely\n",
                                       rank, expert_idx, local_expert_idx);
                                timeout_counter = 0;
                            }
                        }
                    }
                    #ifdef DEEPEP_VERBOSE_DEBUG
                    printf("[FLAG WAIT DONE] rank=%d, expert_idx=%d, local_idx=%d, flag_value=%d, blockIdx.x=%d\n",
                           rank, expert_idx, local_expert_idx, ld_acquire_sys_global((const int*)flag_ptr), blockIdx.x);
                    #endif
                }
            }
        }
    }  // End of if (expert_idx < num_experts)

    // Proper grid-wide synchronization
    // All warps must complete their work before grid sync
    __syncthreads();

    // All threads participate in grid sync (collective operation)
    cg::this_grid().sync();

    // Additional system-wide memory fence for RDMA visibility
    // All threads must execute the fence for consistency
    __threadfence_system();

    // Counter for missing remote expert results
    __shared__ int missing_remote_experts;
    if (thread_id == 0) {
        missing_remote_experts = 0;
    }
    __syncthreads();

    // Reduce tokens with FP8 cast

    EP_DEVICE_ASSERT(num_topk <= 32);
    EP_DEVICE_ASSERT(actual_num_int4_per_data <= num_threads);
    EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerInt4) == 0, "Invalid vectorization");

    // Validate safe_num_topk before use to prevent memory corruption
    bool is_corrupted = (safe_num_topk <= 0 || safe_num_topk > 32);
    if (is_corrupted) {
        if (thread_id == 0 && sm_id == 0) {
            printf("[FATAL] safe_num_topk corrupted: %d (0x%x), original param=%d\n",
                   safe_num_topk, (unsigned int)safe_num_topk, num_topk);
            printf("[FATAL] Common corruption patterns:\n");
            printf("  0x9DC79A28 = freed memory\n");
            printf("  0xF49E0AA8 = uninitialized\n");
            printf("  0xEC01D228 = stack corruption\n");
            printf("[FATAL] Other params: num_experts=%d, num_ranks=%d, num_combined_tokens=%d\n",
                   safe_num_experts, safe_num_ranks, safe_num_combined_tokens);
        }
        // Cannot return early - must participate in grid sync
        // Set safe values to prevent crash
        safe_num_topk = 1;  // Minimum valid value
    }

    // Use actual size and ensure thread is within bounds
    // Process tokens even if skip_recv_phase is true to maintain NVSHMEM collective consistency
    if (thread_id < num_threads) {
        for (int token_idx = sm_id; token_idx < safe_num_combined_tokens; token_idx += num_sms) {
            // token_idx is the global token index we're combining results for

            // Ensure token_idx is within bounds
            if (token_idx >= safe_num_combined_tokens) {
                break;
            }

            // Additional safety check
            if (token_idx < 0) {
                if (thread_id == 0) {
                    printf("[ERROR] Invalid token_idx=%d, sm_id=%d, num_sms=%d\n", token_idx, sm_id, num_sms);
                }
                break;
            }

            // Read top-k indices and weights
            int reg_topk_idx[kNumMaxTopk];
            float reg_topk_weights[kNumMaxTopk];

            // Initialize arrays to safe values
            #pragma unroll
            for (int i = 0; i < kNumMaxTopk; ++i) {
                reg_topk_idx[i] = -1;
                reg_topk_weights[i] = 0.0f;
            }

            // Bounds check before reading topk arrays
            const size_t topk_offset = token_idx * safe_num_topk;
            if (topk_idx == nullptr || topk_weights == nullptr) {
                if (thread_id == 0) {
                    printf("[ERROR] Null pointer: topk_idx=%p, topk_weights=%p\n", topk_idx, topk_weights);
                }
            } else {
                #pragma unroll
                for (int i = 0; i < safe_num_topk; ++ i) {
                    reg_topk_idx[i] = static_cast<int>(__ldg(topk_idx + topk_offset + i));
                    reg_topk_weights[i] = __ldg(topk_weights + topk_offset + i);
                }
            }

            // Stride over int4 chunks to cover full hidden size even when blockDim.x < actual_num_int4_per_data
            for (int int4_idx = thread_id; int4_idx < actual_num_int4_per_data; int4_idx += num_threads) {
                float combined_values[kNumElemsPerInt4] = {0.0f};

                // Accumulate contributions from local experts only
                for (int i = 0; i < safe_num_topk; ++i) {
                    const int src_expert_idx_local = reg_topk_idx[i];
                    if (src_expert_idx_local < 0) continue;

                    const int src_rank_local = src_expert_idx_local / safe_num_local_experts;
                    const int src_local_expert_idx = src_expert_idx_local % safe_num_local_experts;

                    // Only local experts contribute here; remote handled by reduction
                    if (src_rank_local == rank && layout_range != nullptr) {
                        // Map [expert, rank, slot] -> original token id
                        const int* local_src_info =
                            src_info + src_local_expert_idx * safe_num_ranks * safe_num_max_dispatch_tokens_per_rank;
                        for (int from_rank = 0; from_rank < num_ranks; ++from_rank) {
                            auto layout = __ldg(layout_range + src_local_expert_idx * num_ranks + from_rank);
                            int num_tokens_from_rank, offset_in_packed;
                            unpack2(layout, num_tokens_from_rank, offset_in_packed);
                            for (int slot = 0; slot < num_tokens_from_rank; ++slot) {
                                const int packed_slot_idx = offset_in_packed + slot;
                                if (packed_slot_idx >= safe_num_max_dispatch_tokens_per_rank) continue;

                                // Check ownership: does this slot belong to current token_idx?
                                const int src_token_id = __ldg(
                                    local_src_info + from_rank * safe_num_max_dispatch_tokens_per_rank + packed_slot_idx);
                                if (src_token_id != token_idx) continue;

                                // Only now load and accumulate
                                const int4* packed_x_ptr = static_cast<const int4*>(x)
                                    + src_local_expert_idx * safe_num_ranks * safe_num_max_dispatch_tokens_per_rank * actual_num_int4_per_data
                                    + (from_rank * safe_num_max_dispatch_tokens_per_rank + packed_slot_idx) * actual_num_int4_per_data;

                                const size_t buffer_offset = packed_x_ptr + int4_idx - static_cast<const int4*>(x);
                                const size_t max_buffer_size = static_cast<size_t>(safe_num_local_experts) * safe_num_ranks * safe_num_max_dispatch_tokens_per_rank * actual_num_int4_per_data;
                                int4 x_vec = (buffer_offset >= max_buffer_size) ? make_int4(0,0,0,0) : ld_nc_global(packed_x_ptr + int4_idx);
                                const hip_bfloat16* x_bf16 = reinterpret_cast<const hip_bfloat16*>(&x_vec);
                                #pragma unroll
                                for (int j = 0; j < kNumElemsPerInt4; ++j) {
                                    combined_values[j] += static_cast<float>(x_bf16[j]) * reg_topk_weights[i];
                                }
                                // Pure EP: per expert per rank at most one slot matches → stop scanning slots for this rank
                                break;
                            }
                        }
                    }
                }

                // Write out this int4 chunk
                if (!skip_recv_phase) {
                    if (token_idx < safe_num_combined_tokens) {
                        if (is_pure_ep_mode) {
                            // Pure EP: keep FP32 values intact; do not perform BF16 in-place conversion
                            const int elements_per_token = actual_num_int4_per_data * kNumElemsPerInt4;
                            float* token_fp32_ptr = reinterpret_cast<float*>(fp32_workspace) + token_idx * elements_per_token;
                            #pragma unroll
                            for (int j = 0; j < kNumElemsPerInt4; ++j) {
                                token_fp32_ptr[int4_idx * kNumElemsPerInt4 + j] = combined_values[j];
                            }
                        } else {
                            // Non-Pure EP: pack 8 bf16 into one int4 and write to output buffer
                            int4 packed_bf16;
                            hip_bfloat16* packed_ptr = reinterpret_cast<hip_bfloat16*>(&packed_bf16);
                            #pragma unroll
                            for (int j = 0; j < kNumElemsPerInt4; ++j) {
                                packed_ptr[j] = static_cast<hip_bfloat16>(combined_values[j]);
                            }
                            int4* combined_x_ptr = static_cast<int4*>(combined_x) + token_idx * actual_num_int4_per_data + int4_idx;
                            *combined_x_ptr = packed_bf16;
                        }
                    } else {
                        if (thread_id == 0) {
                            printf("[ERROR] Write out of bounds! token_idx=%d, int4_idx=%d\n", token_idx, int4_idx);
                        }
                    }
                } else if (is_pure_ep_mode) {
                    // Do not zero-out FP32 workspace on skip_recv_phase; keep previous partials for NVSHMEM reduction
                    // No-op to avoid tail sections being overwritten to zero by non-owner ranks
                }
            } // end int4 stride loop
        }
    }

    // Perform NVSHMEM reduction for Pure EP mode
    if (is_pure_ep_mode) {
        // Grid-wide sync before reduction to ensure all ranks have written their partial results
        cg::this_grid().sync();

        // If this PE skipped recv phase, its FP32 workspace may contain stale data from previous steps.
        // Zero the entire FP32 workspace so that this PE contributes pure zeros to the reduction.
        if (blockIdx.x == 0 && skip_recv_phase) {
            const int elements_per_token_zero = actual_num_int4_per_data * kNumElemsPerInt4;
            const int max_tokens_zero = safe_num_combined_tokens;
            for (int token_idx_zero = 0; token_idx_zero < max_tokens_zero; ++token_idx_zero) {
                float* p = reinterpret_cast<float*>(fp32_workspace) + token_idx_zero * elements_per_token_zero;
                for (int e = thread_id; e < elements_per_token_zero; e += blockDim.x) {
                    p[e] = 0.0f;
                }
                __syncthreads();
            }
        }

        // Ensure all blocks see a clean FP32 workspace state before entering collective reduction
        cg::this_grid().sync();

        // Use the agreed, safe number of tokens to reduce across PEs
        // Must exactly match the number of tokens combined in this pass
        EP_DEVICE_ASSERT(fp32_workspace != nullptr);
        const int max_combined_tokens = safe_num_combined_tokens;  // equals num_combined_tokens

        // Only block 0 performs NVSHMEM reduction
        // All PEs must have exactly one block 0 that calls the collective
        if (blockIdx.x == 0) {
            const int elements_per_token = actual_num_int4_per_data * kNumElemsPerInt4;
            EP_DEVICE_ASSERT(elements_per_token > 0 && max_combined_tokens > 0);

            // Zero-fill FP32 workspace for tokens beyond local num_combined_tokens
            // to avoid stale data contributing to NVSHMEM reduction.
            // Each PE may have different num_combined_tokens; the reduction loops up to
            // max_combined_tokens for collective consistency, so invalid local tokens
            // must be explicitly set to 0.
            {
                // All threads in block 0 participate to parallelize zero-fill
                for (int token_idx = num_combined_tokens; token_idx < max_combined_tokens; ++token_idx) {
                    float* token_fp32_ptr = reinterpret_cast<float*>(fp32_workspace) + token_idx * elements_per_token;
                    for (int elem = thread_id; elem < elements_per_token; elem += blockDim.x) {
                        token_fp32_ptr[elem] = 0.0f;
                    }
                    __syncthreads();
                }
                // Removed grid-wide sync here; global syncs occur at 1807, 1887, 1915
            }

            // Process ALL tokens up to max_combined_tokens to ensure consistent participation
            for (int token_idx = 0; token_idx < max_combined_tokens; token_idx++) {
                float* token_fp32_ptr = reinterpret_cast<float*>(fp32_workspace) + token_idx * elements_per_token;

                // Check if this is a valid token for this PE
                bool is_valid_token = (token_idx < num_combined_tokens);

                // Log pre-reduction values for valid tokens only
                if (thread_id == 0 && token_idx < 2 && is_valid_token) {
                    printf("[NVSHMEM_REDUCE] rank=%d, block=0, token=%d, pre-reduce: %.6f, %.6f, %.6f\n",
                           rank, token_idx, token_fp32_ptr[0], token_fp32_ptr[1], token_fp32_ptr[2]);
                }

                // Process in chunks to avoid memory issues
                const int chunk_size = 256;  // Process 256 floats at a time

                // Calculate number of chunks (same for all PEs)
                const int num_chunks = (elements_per_token + chunk_size - 1) / chunk_size;

                for (int chunk_idx = 0; chunk_idx < num_chunks; chunk_idx++) {
                    const int chunk_start = chunk_idx * chunk_size;
                    const int chunk_end = min(chunk_start + chunk_size, elements_per_token);
                    const int chunk_elems = chunk_end - chunk_start;

                    // For invalid tokens, use dummy data (zeros already in workspace)
                    // This ensures all PEs call the collective the same number of times

                    // All threads in block 0 must participate in the collective
                    // Use block-level NVSHMEM reduction
                    nvshmemx_float_sum_reduce_block(NVSHMEM_TEAM_WORLD,
                                                  token_fp32_ptr + chunk_start,  // destination
                                                  token_fp32_ptr + chunk_start,  // source
                                                  chunk_elems);                  // count

                    // Sync after each chunk to ensure completion
                    __syncthreads();
                }

                // Log post-reduction values for valid tokens only
                if (thread_id == 0 && token_idx < 2 && is_valid_token) {
                    printf("[NVSHMEM_REDUCE] rank=%d, block=0, token=%d, post-reduce: %.6f, %.6f, %.6f\n",
                           rank, token_idx, token_fp32_ptr[0], token_fp32_ptr[1], token_fp32_ptr[2]);
                }
            }
        }

        // All blocks sync here to ensure block 0 has completed reduction
        cg::this_grid().sync();

        // Now ALL blocks participate in converting FP32 back to BF16
        // Only convert valid tokens (up to num_combined_tokens)
        const int elements_per_token = actual_num_int4_per_data * kNumElemsPerInt4;
        const int tokens_per_block = (num_combined_tokens + gridDim.x - 1) / gridDim.x;
        const int start_token = blockIdx.x * tokens_per_block;
        const int end_token = min(start_token + tokens_per_block, num_combined_tokens);

        // Each block converts its assigned tokens from FP32 to BF16
        for (int token_idx = start_token; token_idx < end_token; token_idx++) {
            float* token_fp32_ptr = reinterpret_cast<float*>(fp32_workspace) + token_idx * elements_per_token;

            // Each thread handles its portion
            const int elems_per_thread = (elements_per_token + blockDim.x - 1) / blockDim.x;
            const int thread_start = thread_id * elems_per_thread;
            const int thread_end = min(thread_start + elems_per_thread, elements_per_token);

            for (int elem_idx = thread_start; elem_idx < thread_end; elem_idx++) {
                hip_bfloat16* bf16_ptr = reinterpret_cast<hip_bfloat16*>(combined_x) +
                                       token_idx * elements_per_token + elem_idx;
                *bf16_ptr = static_cast<hip_bfloat16>(token_fp32_ptr[elem_idx]);
            }

            __syncthreads();
        }

        // Final grid sync to ensure all conversions complete
        cg::this_grid().sync();
    }

    // Reset flags for next iteration using atomic decrement
    // This handles the race condition where multiple ranks may have incremented the flag
    // Each block handles its assigned expert
    if (thread_id == 0 && responsible_expert_idx < num_experts) {
        const auto expert_idx = responsible_expert_idx;
        const auto expert_rank = expert_idx / num_local_experts;
        if (expert_rank == rank) {
            // This expert belongs to this rank, handle it
            const auto local_expert_idx = expert_idx % num_local_experts;

            // Additional bounds check
            if (local_expert_idx < num_local_experts) {
                // Check if we actually waited for this expert (i.e., it received data)
                bool did_wait_for_expert = false;
                if (layout_range != nullptr) {
                    for (int src_rank = 0; src_rank < num_ranks; src_rank++) {
                        auto layout = __ldg(layout_range + local_expert_idx * num_ranks + src_rank);
                        int num_tokens_from_rank, offset;
                        unpack2(layout, num_tokens_from_rank, offset);
                        if (num_tokens_from_rank > 0) {
                            did_wait_for_expert = true;
                            break;
                        }
                    }
                }

                if (did_wait_for_expert) {
                    // More careful flag handling to avoid race conditions
                    // First ensure all reads are complete
                    __threadfence_system();
                    asm volatile("membar.sys;");

                    // Count actual tokens received from all ranks
                    int total_received = 0;
                    for (int src_rank = 0; src_rank < num_ranks; src_rank++) {
                        auto layout = __ldg(layout_range + local_expert_idx * num_ranks + src_rank);
                        int num_tokens_from_rank, offset;
                        unpack2(layout, num_tokens_from_rank, offset);
                        total_received += num_tokens_from_rank;
                    }

                    // Only decrement by the actual number of senders
                    // This prevents over-decrementing if multiple ranks sent data
                    int expected_decrements = 0;
                    for (int src_rank = 0; src_rank < num_ranks; src_rank++) {
                        if (src_rank != rank) {  // Only remote ranks set flags
                            auto layout = __ldg(layout_range + local_expert_idx * num_ranks + src_rank);
                            int num_tokens_from_rank, offset;
                            unpack2(layout, num_tokens_from_rank, offset);
                            if (num_tokens_from_rank > 0) {
                                expected_decrements++;
                            }
                        }
                    }

                    // Safe atomic update
                    if (expected_decrements > 0) {
                        atomicSub(&rdma_recv_flag[local_expert_idx], expected_decrements);
                        __threadfence_system();

                        #ifdef DEEPEP_VERBOSE_DEBUG
                        int final_value = rdma_recv_flag[local_expert_idx];
                        printf("[FLAG DECREMENT] rank=%d decremented flag for local_expert=%d by %d, new value=%d\n",
                               rank, local_expert_idx, expected_decrements, final_value);
                        #endif
                    }
                }
            } else {
                printf("[ERROR] Invalid local_expert_idx=%d, expert_idx=%d, num_local_experts=%d\n",
                       local_expert_idx, expert_idx, num_local_experts);
            }
        } // End if (expert_rank == rank)
    }

    // Report missing remote experts at the end
    __syncthreads();
    if (thread_id == 0 && missing_remote_experts > 0) {
        printf("[CRITICAL] rank=%d: %d remote expert results are missing!\n",
               rank, missing_remote_experts);
        printf("  This causes incorrect output. NVSHMEM+NCCL conflict prevents AllReduce.\n");
        printf("  Need to implement NVSHMEM-based aggregation in the kernel.\n");
    }
}

void combine(void* combined_x,
             void* fp32_workspace,  // Separate FP32 workspace for NVSHMEM reduction
             void* rdma_recv_x, int* rdma_recv_flag, void* rdma_send_x,
             ExpertSyncInfo* expert_sync_info_buffer,  // Expert synchronization info
             const void* x, const int64_t* topk_idx, const float* topk_weights,
             const int* src_info, const int64_t* layout_range,
             int64_t* combine_wait_recv_cost_stats,
             int* next_clean, int num_next_clean_int,
             int num_combined_tokens, int hidden, int num_max_dispatch_tokens_per_rank,
             int num_topk, int num_experts, int rank, int num_ranks,
             bool use_logfmt,
             void* workspace, int num_device_sms,
             hipStream_t stream, int phases, bool zero_copy) {
    // Add detailed parameter validation
    EP_HOST_ASSERT(combined_x != nullptr);
    EP_HOST_ASSERT(x != nullptr);
    EP_HOST_ASSERT(topk_idx != nullptr);
    EP_HOST_ASSERT(topk_weights != nullptr);
    EP_HOST_ASSERT(num_combined_tokens > 0);
    EP_HOST_ASSERT(hidden > 0 && hidden % 128 == 0);
    EP_HOST_ASSERT(num_experts > 0 && num_experts % num_ranks == 0);
    EP_HOST_ASSERT(num_ranks > 0);
    EP_HOST_ASSERT(num_topk > 0 && num_topk <= 9);

    // Set grid size to num_experts to ensure all experts are processed
    // Each block will handle one expert
    const int grid_size = num_experts;

    // Calculate warps based on device SM count, not experts
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = min(32 / num_warp_groups, 32);
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);

    const auto num_warps = min(num_warp_groups * num_warps_per_group, 32);

    // Check workspace
    auto atomic_clean_flag = static_cast<int*>(workspace);
    EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
    EP_HOST_ASSERT(num_topk <= 9);

#define COMBINE_LAUNCH_CASE(hidden) { \
auto combine_func = combine<hidden, 9>; \
LAUNCH_KERNEL(&cfg, combine_func, \
              combined_x, \
              fp32_workspace, \
              rdma_recv_x, rdma_recv_flag, rdma_send_x, \
              x, topk_idx, topk_weights, src_info, layout_range, \
              combine_wait_recv_cost_stats, \
              next_clean, num_next_clean_int, \
              atomic_clean_flag, \
              expert_sync_info_buffer, \
              num_combined_tokens, hidden, num_topk, \
              num_max_dispatch_tokens_per_rank, \
              num_experts, rank, num_ranks, \
              num_warp_groups, num_warps_per_group, \
              phases, zero_copy); } break

    // Setup launch configuration
    // Ensure threads per block cover all int4 chunks of hidden (hidden/8 for bf16)
    const int required_threads_int4 = hidden / 8;  // bf16: 8 elems per int4
    const int threads_per_block = (num_warps * 32 > required_threads_int4) ? (num_warps * 32) : required_threads_int4;
    SETUP_LAUNCH_CONFIG(grid_size, threads_per_block, stream);  // Use grid_size instead of num_experts
    SWITCH_HIDDEN(COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

} // namespace internode_ll

} // namespace deep_ep
